
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (void)
{
    printf("Hello world from GPU!\n");
}

int main()
{
    printf("Hello world from CPU!\n");

    helloFromGPU <<<10, 10>>>();
    hipDeviceReset();

    return 0;
}